#include "hip/hip_runtime.h"
/*--------------------------------*- BHgpu -*----------------*---------------*\
| #####   ##  ##                |                            | Version 1.5    |
| ##  ##  ##  ##   ####  ##  ## |  BHgpu: Barnes-Hut method  | 2023/08/29     |
| #####   ######  ##     ##  ## |  for 2D vortex particles   *----------------*
| ##  ##  ##  ##  ##     ##  ## |  Open Source Code                           |
| #####   ##  ##   ####   ####  |  https://www.github.com/vortexmethods/fastm |
|                                                                             |
| Copyright (C) 2020-2023 I. Marchevsky, E. Ryatina, A. Kolganova             |
| Copyright (C) 2013, Texas State University-San Marcos. All rights reserved. |
*-----------------------------------------------------------------------------*
| File name: SummKer_n.cu                                                     |
| Info: Source code of BHgpu                                                  |
|                                                                             |
| This file is part of BHgpu.                                                 |
| BHcu is free software: you can redistribute it and/or modify it             |
| under the terms of the GNU General Public License as published by           |
| the Free Software Foundation, either version 3 of the License, or           |
| (at your option) any later version.                                         |
|                                                                             |
| BHcu is distributed in the hope that it will be useful, but WITHOUT         |
| ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or       |
| FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License       |
| for more details.                                                           |
|                                                                             |
| You should have received a copy of the GNU General Public License           |
| along with BHgpu.  If not, see <http://www.gnu.org/licenses/>.              |
\*---------------------------------------------------------------------------*/

/*!
\file
\brief Сдвиг мультипольных моментов для схемы с order = 2
\author Марчевский Илья Константинович
\author Рятина Евгения Павловна
\author Колганова Александра Олеговна
\version 1.5
\date 29 августа 2023 г.
*/


__global__ 
__launch_bounds__(THREADS3, FACTOR3)
void SummarizationKernel2_2(
    const int nnodesd, const int nbodiesd,
    const int2* __restrict Mchildd,
    volatile int* __restrict massd,
    const int order, real2* __restrict momsd,  //momsd  - без volatile
    const double* __restrict vtxd, int objectType, const int* __restrict MmortonCodesIdxd,
    const real2* __restrict Mposd, const int* __restrict MindexSortd, const int* __restrict MindexSortTd
)
{
    register int i, j, k, ch, inc, flag;

    register real2 mom0;
    register real2 mom1;

    register real2 cen, dr;

    register int m, cm;

    inc = blockDim.x * gridDim.x;
    k = ((nnodesd - (nbodiesd - 1)) & (-WARPSIZE)) + threadIdx.x + blockIdx.x * blockDim.x;  // align to warp size

    if (k < (nnodesd - (nbodiesd - 1)))
        k += inc;

    //MortonTree:
    // 0 1 2 ... (nb-2) x (nb+0) (nb+1) (nb+2) ... (nb+(nb-1))
    // ----------------   -----------------------------------
    //      cells                         bodies

    //Martin's tree:
    // 0 1 2 ... (nb-1) x x x x (nn-(nb-1)) ... (nn-2) (nn-1)
    // ----------------          ----------------------------
    //      bodies                 sorted and reversed cells

    flag = 0;
    j = 0;
    // iterate over all cells assigned to thread
    while (k < nnodesd)
    {
        if (massd[nnodesd - 1 - k] >= 0)
        {
            k += inc;
        }
        else
        {
            j = 2;
            for (i = 0; i < 2; i++) {

                //computation of child[k*2+i]
                register const int srt = MindexSortd[(nnodesd - 1) - k];
                int chd = i * Mchildd[srt].y + (1-i) * Mchildd[srt].x;   // i==0 => .x;  i==1 => .y
                ch = (chd >= nbodiesd) ? chd - nbodiesd : (nnodesd - 1) - MindexSortTd[chd];

                if ((chd >= nbodiesd) || (massd[nnodesd - 1 - ch] >= 0))
                    j--;
            }

            if (j == 0) {
                // all children are ready
                const int kch = ((nnodesd - 1) - k) * order;
                cm = 0;

                const register int sortedCell = MindexSortd[(nnodesd - 1) - k];

                cen = Mposd[sortedCell];
                const int2 chdPair = Mchildd[sortedCell];

                for (i = 0; i < 2; i++)
                {
                    //computation of ch = child[k*2+i]
                    const int chd = i * chdPair.y + (1-i) * chdPair.x;
                    if (chd >= nbodiesd)
                    {
                         ch = chd - nbodiesd;
                         const register int sortedBody = MmortonCodesIdxd[ch];
                         if (objectType == 0)
                         {
                              mom0 = real2{ vtxd[sortedBody*3+2], (real)0 };
                              mom1 = real2{ 0, 0 };
                              dr = real2{vtxd[sortedBody*3+0], vtxd[sortedBody*3+1]} - cen;
                              m = 1;
                         } //objectType==0
                         if ((objectType == 1) || (objectType == 2) || (objectType == -1) || (objectType == -2))
                         {
                              real2 panBegin, panEnd;
                              panBegin = real2{vtxd[sortedBody * 12 + 2], vtxd[sortedBody * 12 + 3]};
                              panEnd = real2{vtxd[sortedBody * 12 + 4], vtxd[sortedBody * 12 + 5]};

                              real2 rcur;
                              rcur = multz(0.5 * (panEnd - panBegin), 0.5 * (panEnd - panBegin));
                              real gam;
                              switch (objectType)
                              {
                              case 1:
                              case 2:
                                  gam = vtxd[sortedBody * 12 + 6] + vtxd[sortedBody * 12 + 7];
                                  break;

                              case -1:
                              case -2:
                                  gam = vtxd[sortedBody * 12 + 8];
                                  break;
                              };

                              mom1 = real2{ 0, 0 };
                              mom0 = real2{ gam, (real)0 };


                         if ((objectType == 2) || (objectType == -2))
                         {
                              real gamLin;

                              switch (objectType)
                              {
                              case 2:
                                  gamLin = vtxd[sortedBody * 12 + 9] + vtxd[sortedBody * 12 + 10];
                                  break;
                              case -2:
                                  gamLin = vtxd[sortedBody * 12 + 11];
                                  break;
                              };

                              rcur = 0.5 * (panEnd - panBegin);
                              mom1 = gamLin * (0.5 / 3) * rcur; 
                         }

                              dr = real2{vtxd[sortedBody*12+0], vtxd[sortedBody*12+1]} - cen;
                              m = 1;
                         } //objectType==1
                    }
                    else
                    {
                         register const int srtT = MindexSortTd[chd];
                         ch = (nnodesd - 1) - srtT;
                         const int nch = srtT * order;
                         mom0 = real2{ momsd[nch + 0].x, (real)0 };
                         mom1 = momsd[nch + 1];
                         //for (int s = 1; s < order; ++s)
                         //    mom[s] = momsd[ch * order + s];

                         dr = Mposd[chd] - cen;
                         m = massd[nnodesd - 1 - ch];
                     }
                     // add child's contribution
                     momsd[kch + 0].x += mom0.x;

                     register real2 momh1 = mom1;

                     //for (int p = 1; p < order; ++p)
                     //    momh[p] = mom[p];

                     real2 z = dr;

                     momh1 += multz(mom0, z);


                     //for (int s = 1; s < order; ++s)
                     //{
                     //    for (int p = s; p < order; ++p)
                     //        momh[p] += binomCft[p * order + s] * multz(mom[p - s], z);
                     //    z = multz(z, dr);
                     //}

                     momsd[kch + 1] += momh1;

                     //for (int p = 1; p < order; ++p)
                     //    momsd[k * (order)+p] += momh[p];

                     cm += m;
                }
                flag = 1;
            }
        }
        __threadfence();

        if (flag != 0) {
            massd[nnodesd - 1 - k] = cm;
            k += inc;
            flag = 0;
        }
    }
}