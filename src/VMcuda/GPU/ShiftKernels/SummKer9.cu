#include "hip/hip_runtime.h"
/*--------------------------------*- BHgpu -*----------------*---------------*\
| #####   ##  ##                |                            | Version 1.5    |
| ##  ##  ##  ##   ####  ##  ## |  BHgpu: Barnes-Hut method  | 2023/08/29     |
| #####   ######  ##     ##  ## |  for 2D vortex particles   *----------------*
| ##  ##  ##  ##  ##     ##  ## |  Open Source Code                           |
| #####   ##  ##   ####   ####  |  https://www.github.com/vortexmethods/fastm |
|                                                                             |
| Copyright (C) 2020-2023 I. Marchevsky, E. Ryatina, A. Kolganova             |
| Copyright (C) 2013, Texas State University-San Marcos. All rights reserved. |
*-----------------------------------------------------------------------------*
| File name: SummKer_n.cu                                                     |
| Info: Source code of BHgpu                                                  |
|                                                                             |
| This file is part of BHgpu.                                                 |
| BHcu is free software: you can redistribute it and/or modify it             |
| under the terms of the GNU General Public License as published by           |
| the Free Software Foundation, either version 3 of the License, or           |
| (at your option) any later version.                                         |
|                                                                             |
| BHcu is distributed in the hope that it will be useful, but WITHOUT         |
| ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or       |
| FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License       |
| for more details.                                                           |
|                                                                             |
| You should have received a copy of the GNU General Public License           |
| along with BHgpu.  If not, see <http://www.gnu.org/licenses/>.              |
\*---------------------------------------------------------------------------*/

/*!
\file
\brief Сдвиг мультипольных моментов для схемы с order = 9
\author Марчевский Илья Константинович
\author Рятина Евгения Павловна
\author Колганова Александра Олеговна
\version 1.5
\date 29 августа 2023 г.
*/


__global__ 
__launch_bounds__(THREADS3, FACTOR3)
void SummarizationKernel2_9(
    const int nnodesd, const int nbodiesd,
    const int2* __restrict Mchildd,
    volatile int* __restrict massd,
    const int order, real2* __restrict momsd,  //momsd  - без volatile
    const double* __restrict vtxd, int objectType, const int* __restrict MmortonCodesIdxd,
    const real2* __restrict Mposd, const int* __restrict MindexSortd, const int* __restrict MindexSortTd
)
{
    register int i, j, k, ch, inc, flag;

    register real2 mom0;
    register real2 mom1;
    register real2 mom2;
    register real2 mom3;
    register real2 mom4;
    register real2 mom5;
    register real2 mom6;
    register real2 mom7;
    register real2 mom8;

    register real2 cen, dr;

    register int m, cm;

    inc = blockDim.x * gridDim.x;
    k = ((nnodesd - (nbodiesd - 1)) & (-WARPSIZE)) + threadIdx.x + blockIdx.x * blockDim.x;  // align to warp size

    if (k < (nnodesd - (nbodiesd - 1)))
        k += inc;

    //MortonTree:
    // 0 1 2 ... (nb-2) x (nb+0) (nb+1) (nb+2) ... (nb+(nb-1))
    // ----------------   -----------------------------------
    //      cells                         bodies

    //Martin's tree:
    // 0 1 2 ... (nb-1) x x x x (nn-(nb-1)) ... (nn-2) (nn-1)
    // ----------------          ----------------------------
    //      bodies                 sorted and reversed cells

    flag = 0;
    j = 0;
    // iterate over all cells assigned to thread
    while (k < nnodesd)
    {
        if (massd[nnodesd - 1 - k] >= 0)
        {
            k += inc;
        }
        else
        {
            j = 2;
            for (i = 0; i < 2; i++) {

                //computation of child[k*2+i]
                register const int srt = MindexSortd[(nnodesd - 1) - k];
                int chd = i * Mchildd[srt].y + (1-i) * Mchildd[srt].x;   // i==0 => .x;  i==1 => .y
                ch = (chd >= nbodiesd) ? chd - nbodiesd : (nnodesd - 1) - MindexSortTd[chd];

                if ((chd >= nbodiesd) || (massd[nnodesd - 1 - ch] >= 0))
                    j--;
            }

            if (j == 0) {
                // all children are ready
                const int kch = ((nnodesd - 1) - k) * order;
                cm = 0;

                const register int sortedCell = MindexSortd[(nnodesd - 1) - k];

                cen = Mposd[sortedCell];
                const int2 chdPair = Mchildd[sortedCell];

                for (i = 0; i < 2; i++)
                {
                    //computation of ch = child[k*2+i]
                    const int chd = i * chdPair.y + (1-i) * chdPair.x;
                    if (chd >= nbodiesd)
                    {
                         ch = chd - nbodiesd;
                         const register int sortedBody = MmortonCodesIdxd[ch];
                         if (objectType == 0)
                         {
                              mom0 = real2{ vtxd[sortedBody*3+2], (real)0 };
                              mom1 = mom2 = mom3 = mom4 = mom5 = mom6 = mom7 = mom8 = real2{ 0, 0 };
                              dr = real2{vtxd[sortedBody*3+0], vtxd[sortedBody*3+1]} - cen;
                              m = 1;
                         } //objectType==0
                         if ((objectType == 1) || (objectType == 2) || (objectType == -1) || (objectType == -2))
                         {
                              real2 panBegin, panEnd;
                              panBegin = real2{vtxd[sortedBody * 12 + 2], vtxd[sortedBody * 12 + 3]};
                              panEnd = real2{vtxd[sortedBody * 12 + 4], vtxd[sortedBody * 12 + 5]};

                              real2 rcur, rd2Pow;
                              rcur = rd2Pow = multz(0.5 * (panEnd - panBegin), 0.5 * (panEnd - panBegin));
                              real gam;
                              switch (objectType)
                              {
                              case 1:
                              case 2:
                                  gam = vtxd[sortedBody * 12 + 6] + vtxd[sortedBody * 12 + 7];
                                  break;

                              case -1:
                              case -2:
                                  gam = vtxd[sortedBody * 12 + 8];
                                  break;
                              };

                              mom1 = mom3 = mom5 = mom7 = real2{ 0, 0 };
                              mom0 = real2{ gam, (real)0 };

                              mom2 = (gam / 3) * rcur; 
                              rcur = multz(rcur, rd2Pow);
                              mom4 = (gam / 5) * rcur; 
                              rcur = multz(rcur, rd2Pow);
                              mom6 = (gam / 7) * rcur; 
                              rcur = multz(rcur, rd2Pow);
                              mom8 = (gam / 9) * rcur; 
                         if ((objectType == 2) || (objectType == -2))
                         {
                              real gamLin;

                              switch (objectType)
                              {
                              case 2:
                                  gamLin = vtxd[sortedBody * 12 + 9] + vtxd[sortedBody * 12 + 10];
                                  break;
                              case -2:
                                  gamLin = vtxd[sortedBody * 12 + 11];
                                  break;
                              };

                              rcur = 0.5 * (panEnd - panBegin);
                              mom1 = gamLin * (0.5 / 3) * rcur; 
                              rcur = multz(rcur, rd2Pow);
                              mom3 = gamLin * (0.5 / 5) * rcur; 
                              rcur = multz(rcur, rd2Pow);
                              mom5 = gamLin * (0.5 / 7) * rcur; 
                              rcur = multz(rcur, rd2Pow);
                              mom7 = gamLin * (0.5 / 9) * rcur; 
                         }

                              dr = real2{vtxd[sortedBody*12+0], vtxd[sortedBody*12+1]} - cen;
                              m = 1;
                         } //objectType==1
                    }
                    else
                    {
                         register const int srtT = MindexSortTd[chd];
                         ch = (nnodesd - 1) - srtT;
                         const int nch = srtT * order;
                         mom0 = real2{ momsd[nch + 0].x, (real)0 };
                         mom1 = momsd[nch + 1];
                         mom2 = momsd[nch + 2];
                         mom3 = momsd[nch + 3];
                         mom4 = momsd[nch + 4];
                         mom5 = momsd[nch + 5];
                         mom6 = momsd[nch + 6];
                         mom7 = momsd[nch + 7];
                         mom8 = momsd[nch + 8];
                         //for (int s = 1; s < order; ++s)
                         //    mom[s] = momsd[ch * order + s];

                         dr = Mposd[chd] - cen;
                         m = massd[nnodesd - 1 - ch];
                     }
                     // add child's contribution
                     momsd[kch + 0].x += mom0.x;

                     register real2 momh1 = mom1;
                     register real2 momh2 = mom2;
                     register real2 momh3 = mom3;
                     register real2 momh4 = mom4;
                     register real2 momh5 = mom5;
                     register real2 momh6 = mom6;
                     register real2 momh7 = mom7;
                     register real2 momh8 = mom8;

                     //for (int p = 1; p < order; ++p)
                     //    momh[p] = mom[p];

                     real2 z = dr;

                     momh1 += multz(mom0, z);
                     momh2 += 2 * multz(mom1, z);
                     momh3 += 3 * multz(mom2, z);
                     momh4 += 4 * multz(mom3, z);
                     momh5 += 5 * multz(mom4, z);
                     momh6 += 6 * multz(mom5, z);
                     momh7 += 7 * multz(mom6, z);
                     momh8 += 8 * multz(mom7, z);

                     z = multz(z, dr);

                     momh2 += multz(mom0, z);
                     momh3 += 3 * multz(mom1, z);
                     momh4 += 6.0 * multz(mom2, z);
                     momh5 += 10.0 * multz(mom3, z);
                     momh6 += 15.0 * multz(mom4, z);
                     momh7 += 21.0 * multz(mom5, z);
                     momh8 += 28.0 * multz(mom6, z);

                     z = multz(z, dr);

                     momh3 += multz(mom0, z);
                     momh4 += 4 * multz(mom1, z);
                     momh5 += 10.0 * multz(mom2, z);
                     momh6 += 20.0 * multz(mom3, z);
                     momh7 += 35.0 * multz(mom4, z);
                     momh8 += 56.0 * multz(mom5, z);

                     z = multz(z, dr);

                     momh4 += multz(mom0, z);
                     momh5 += 5 * multz(mom1, z);
                     momh6 += 15.0 * multz(mom2, z);
                     momh7 += 35.0 * multz(mom3, z);
                     momh8 += 70.0 * multz(mom4, z);

                     z = multz(z, dr);

                     momh5 += multz(mom0, z);
                     momh6 += 6 * multz(mom1, z);
                     momh7 += 21.0 * multz(mom2, z);
                     momh8 += 56.0 * multz(mom3, z);

                     z = multz(z, dr);

                     momh6 += multz(mom0, z);
                     momh7 += 7 * multz(mom1, z);
                     momh8 += 28.0 * multz(mom2, z);

                     z = multz(z, dr);

                     momh7 += multz(mom0, z);
                     momh8 += 8 * multz(mom1, z);

                     z = multz(z, dr);

                     momh8 += multz(mom0, z);

                     //for (int s = 1; s < order; ++s)
                     //{
                     //    for (int p = s; p < order; ++p)
                     //        momh[p] += binomCft[p * order + s] * multz(mom[p - s], z);
                     //    z = multz(z, dr);
                     //}

                     momsd[kch + 1] += momh1;
                     momsd[kch + 2] += momh2;
                     momsd[kch + 3] += momh3;
                     momsd[kch + 4] += momh4;
                     momsd[kch + 5] += momh5;
                     momsd[kch + 6] += momh6;
                     momsd[kch + 7] += momh7;
                     momsd[kch + 8] += momh8;

                     //for (int p = 1; p < order; ++p)
                     //    momsd[k * (order)+p] += momh[p];

                     cm += m;
                }
                flag = 1;
            }
        }
        __threadfence();

        if (flag != 0) {
            massd[nnodesd - 1 - k] = cm;
            k += inc;
            flag = 0;
        }
    }
}