#include "hip/hip_runtime.h"
/*-------------------------------*- VMcuda -*----------------*---------------*\
| ##  ## ##   ##  ####  #####   |                            | Version 1.12   |
| ##  ## ### ### ##  ## ##  ##  |  VMcuda: VM2D/VM3D Library | 2024/01/14     |
| ##  ## ## # ##    ##  ##  ##  |  Open Source Code          *----------------*
|  ####  ##   ##   ##   ##  ##  |  https://www.github.com/vortexmethods/VM2D  |
|   ##   ##   ## ###### #####   |  https://www.github.com/vortexmethods/VM3D  |
|                                                                             |
| Copyright (C) 2017-2024 Ilia Marchevsky                                     |
*-----------------------------------------------------------------------------*
| File name: cuLib2D.cu                                                       |
| Info: Source code of VMcuda                                                 |
|                                                                             |
| This file is part of VMcuda.                                                |
| VMcuda is free software: you can redistribute it and/or modify it           |
| under the terms of the GNU General Public License as published by           |
| the Free Software Foundation, either version 3 of the License, or           |
| (at your option) any later version.                                         |
|                                                                             |
| VMcuda is distributed in the hope that it will be useful, but WITHOUT       |
| ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or       |
| FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License       |
| for more details.                                                           |
|                                                                             |
| You should have received a copy of the GNU General Public License           |
| along with VMcuda.  If not, see <http://www.gnu.org/licenses/>.             |
\*---------------------------------------------------------------------------*/


/*!
\file
\brief Файл с реализацией функций библиотеки VMcuda для работы с CUDA
\author Марчевский Илья Константинович
\Version 1.12
\date 14 января 2024 г.
*/

#include <iostream>
#include <algorithm>

#include "cuLib2D.cuh"

#include "hip/hip_runtime.h"
#include "Gpudefs.h"

#include <hipsolver.h>


__device__ __constant__ size_t sizeVort;
__device__ __constant__ size_t posR;
__device__ __constant__ size_t posG;

__device__ __constant__ double accelCoeff;

__device__ __constant__ double maxGamma;
__device__ __constant__ double collapseRightBorder;
__device__ __constant__ double collapseScale;

__device__ __constant__ double iDPIminEpsAst2;

__device__ __constant__ int schemeSwitcher;
__device__ __constant__ int currentStep;


#define invdpi (0.15915494309189533576888376337251)
#define pi (3.1415926535897932384626433832795)


void cuAlloc(void** ptr, size_t numBytes)
{
	hipHostAlloc(ptr, numBytes, hipHostMallocDefault);
}

void cuDalloc(void* ptr)
{
	hipError_t err1 = hipHostFree(ptr);

	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (cuDalloc) " << std::endl;
}


__device__ double myAtomicAdd(double* address, double val)
{
	unsigned long long int* address_as_ull = (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;

	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val +	__longlong_as_double(assumed)));

		// Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
	} while (assumed != old);
	return __longlong_as_double(old);
}





__device__ inline double myMax(double x, double y)
{
	return (x > y) ? x : y;
}

__device__ inline double myMin(double x, double y)
{
	return (x > y) ? y : x;
}

__device__ inline int sqr(int x)
{
	return x * x;
}

__device__ inline double sqr(double x)
{
	return x * x;
}


/// \brief Способ сглаживания скорости вихря (вихрь Рэнкина или вихрь Ламба)
__device__ inline double CUboundDenom(double r2, double eps2)
{
#ifndef LAMBVORTEX
	return myMax(r2, eps2);
#else
	if (r2 > eps2)
		return r2;
	else
		return (r2 < 1e-10) ? 1e-10 : r2  / (1.0 - exp(-6.0*r2 / eps2));
#endif
}



//Ниже - ядра (__global__), затем - "обычные" функции (__host__)

__global__ void CU_WakeToZero(size_t nvt, double* vt)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;	

	if (i < nvt)
	{
		vt[i * sizeVort + posR + 0] = 0.0;
		vt[i * sizeVort + posR + 1] = 0.0;
		vt[i * sizeVort + posG] = 0.0;
	}
}


__global__ void CU_calc_conv_epsast(
	size_t npt, double* pt,
	size_t nvt, double* vt,
	size_t nsr, double* sr,
	double eps2,
	double* vel, double* rad,
	size_t nAfls, size_t* nVtxs, double** ptrVtxs,
	bool calcVelo, bool calcRadius)
{	
	__shared__ double shx[CUBLOCK];
	__shared__ double shy[CUBLOCK];
	__shared__ double shg[CUBLOCK];

	size_t i = blockIdx.x * blockDim.x + threadIdx.x;

	double ptx = pt[i*sizeVort + posR + 0];
	double pty = pt[i*sizeVort + posR + 1];

	double velx = 0.0;
	double vely = 0.0;

	double dx, dy, dr2;
	double izn;
		
#ifndef TESTONLYVELO
	double d_1 = 1e+5;
	double d_2 = 1e+5;
	double d_3 = 1e+5;
	double d_0 = 1e+5;
	double dst23, dst12, dst01;
#endif	

	//vortices
	for (size_t j = 0; j < nvt; j += CUBLOCK)
	{		
		shx[threadIdx.x] = vt[(j + threadIdx.x)*sizeVort + posR + 0];
		shy[threadIdx.x] = vt[(j + threadIdx.x)*sizeVort + posR + 1];
		shg[threadIdx.x] = vt[(j + threadIdx.x)*sizeVort + posG + 0];

		__syncthreads();
	
		for (size_t q = 0; q < CUBLOCK; ++q)
		{
			if (j + q < nvt)
			{
				dx = ptx - shx[q];
				dy = pty - shy[q];
				dr2 = dx * dx + dy * dy;

				if (calcVelo)
				{
					izn = shg[q] / myMax(dr2, eps2);// / CUboundDenom(dr2, eps2); //Сглаживать надо!!!
										
					velx -= dy * izn;
					vely += dx * izn;
				}

#ifndef TESTONLYVELO
				if (calcRadius)
				{
					if (d_3 > dr2)
					{
						//if(i == 93)
						//printf("upd: from %d, dr2 = %f\n", (int)(j+q), dr2);

						dst23 = myMin(dr2, d_2);
						d_3 = myMax(dr2, d_2);

						dst12 = myMin(dst23, d_1);
						d_2 = myMax(dst23, d_1);

						dst01 = myMin(dst12, d_0);
						d_1 = myMax(dst12, d_0);
						d_0 = dst01;
					}
				}
#endif
			}
		}
		__syncthreads();
	}
	//if (i == 93)
	//	printf("A: d0,1,2,3: %f, %f, %f, %f\n", d_0, d_1, d_2, d_3);


	//sources
	if (calcVelo)
	{
		for (size_t j = 0; j < nsr; j += CUBLOCK)
		{
			shx[threadIdx.x] = sr[(j + threadIdx.x)*sizeVort + posR + 0];
			shy[threadIdx.x] = sr[(j + threadIdx.x)*sizeVort + posR + 1];
			shg[threadIdx.x] = sr[(j + threadIdx.x)*sizeVort + posG + 0] * accelCoeff;

			__syncthreads();

			for (size_t q = 0; q < CUBLOCK; ++q)
			{
				if (j + q < nsr)
				{
					dx = ptx - shx[q];
					dy = pty - shy[q];
					dr2 = dx * dx + dy * dy;

					izn = shg[q] / CUboundDenom(dr2, eps2); //Сглаживать надо!!!

					velx += dx * izn;
					vely += dy * izn;
				}
			}
			__syncthreads();
		}
	}

	if (calcRadius)
	{
#ifndef TESTONLYVELO
		/*
		size_t nTotVtxs = 0;
		for (size_t p = 0; p < nAfls; ++p)
			nTotVtxs += nVtxs[p];

		//for (size_t p = 0; p < nAfls; ++p)						
		size_t p = 0;
			//for (size_t j = 0; j < nVtxs[p]; j += CUBLOCK)
			for (size_t j = 0; j < nTotVtxs; j += CUBLOCK)
			{
				//if (((&ptrVtxs[p][0] + ((j + threadIdx.x) * sizeVort + posR + 1)) - &ptrVtxs[0][0]) > 4096 * 3)
				//{
				//	//printf("!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!! %d\n", ((&ptrVtxs[p][0] + ((j + threadIdx.x) * sizeVort + posR + 1)) - &ptrVtxs[0][0]));
				//	printf("p = %d, nVtxs[p] = %d, j = %d\n", (int)p, (int)(nVtxs[p]), (int)j);
				//}

				shx[threadIdx.x] = ptrVtxs[p][(j + threadIdx.x)*sizeVort + posR + 0];
				shy[threadIdx.x] = ptrVtxs[p][(j + threadIdx.x)*sizeVort + posR + 1];

				__syncthreads();

				for (size_t q = 0; q < CUBLOCK; ++q)
				{
					if (j + q < nVtxs[p])
					{
						dx = ptx - shx[q];
						dy = pty - shy[q];
						dr2 = dx * dx + dy * dy;


						if (d_3 > dr2)
						{
							dst23 = myMin(dr2, d_2);
							d_3 = myMax(dr2, d_2);

							dst12 = myMin(dst23, d_1);
							d_2 = myMax(dst23, d_1);

							dst01 = myMin(dst12, d_0);
							d_1 = myMax(dst12, d_0);
							d_0 = dst01;
						}
					}
				}
				__syncthreads();
			}
			if (i == 0)
				printf("B: d0,1,2,3: %f, %f, %f, %f\n", d_0, d_1, d_2, d_3);
		*/
#endif
	}
	
	if (i < npt)
	{
		if (calcVelo)
		{
			vel[2 * i + 0] = velx * invdpi;
			vel[2 * i + 1] = vely * invdpi;
		}

		if (calcRadius)
		{
#ifndef TESTONLYVELO
			rad[i] =  1.0 * sqrt((d_1 + d_2 + d_3) * 0.3333333333333333);

			//printf("rad_%d = %f\n", (int)i, rad[i]);

			//rad[i] =  4.0 * sqrt((d_1 + d_2 + d_3) * 0.3333333333333333);
#endif
		}
	}	
}



__global__ void CU_calc_conv_From_Panels(
	size_t npt, double* pt,
	size_t npnl, double* r, 
	double* freegamma, double* freegammalin,
	double* attgamma, double* attgammalin,
	double* attsource, double* attsourcelin,
	double eps2,
	double* vel)
{
	__shared__ double shx[CUBLOCK];
	__shared__ double shy[CUBLOCK];

	__shared__ double shdx[CUBLOCK];
	__shared__ double shdy[CUBLOCK];

	__shared__ double shlen[CUBLOCK];


	__shared__ double shfreegamma[CUBLOCK];
	__shared__ double shattgamma[CUBLOCK];
	__shared__ double shattsource[CUBLOCK];

	__shared__ double shfreegammaLin[CUBLOCK];
	__shared__ double shattgammaLin[CUBLOCK];
	__shared__ double shattsourceLin[CUBLOCK];

	size_t i = blockIdx.x * blockDim.x + threadIdx.x;

	double ptx = pt[i*sizeVort + posR + 0];
	double pty = pt[i*sizeVort + posR + 1];

	double velx = 0.0;
	double vely = 0.0;

	double sx, sy, px, py, s2, p2, alpha, lambda, taux, tauy, u1x, u1y, skos0x, skos0y, skos1x, skos1y;

	//if (i == 0)
	//	printf("Point[0]: (%f, %f)\n", ptx, pty);

	for (size_t j = 0; j < npnl; j += CUBLOCK)
	{
		shx[threadIdx.x] = r[(j + threadIdx.x) * 4 + 0];
		shy[threadIdx.x] = r[(j + threadIdx.x) * 4 + 1];
		
		shdx[threadIdx.x] = r[(j + threadIdx.x) * 4 + 2] - shx[threadIdx.x];
		shdy[threadIdx.x] = r[(j + threadIdx.x) * 4 + 3] - shy[threadIdx.x];

		shlen[threadIdx.x] = sqrt(shdx[threadIdx.x] * shdx[threadIdx.x] + shdy[threadIdx.x] * shdy[threadIdx.x]);

		shfreegamma[threadIdx.x] = freegamma[j + threadIdx.x];
		shattgamma[threadIdx.x] = attgamma[j + threadIdx.x];
		shattsource[threadIdx.x] = attsource[j + threadIdx.x];

		if (schemeSwitcher == 2)
		{
			shfreegammaLin[threadIdx.x] = freegammalin[j + threadIdx.x];
			shattgammaLin[threadIdx.x] = attgammalin[j + threadIdx.x];
			shattsourceLin[threadIdx.x] = attsourcelin[j + threadIdx.x];
		}

		__syncthreads();

		for (size_t q = 0; q < CUBLOCK; ++q)
		{
			if (j + q < npnl)
			{
				sx = ptx - shx[q];
				sy = pty - shy[q];

				px = sx - shdx[q];
				py = sy - shdy[q];

				alpha = atan2(px*sy - py*sx, px*sx + py*sy);
				
				s2 = sx * sx + sy * sy;
				p2 = px * px + py * py;

				if ((s2 > 1e-16) && (p2 > 1e-16)) 
					lambda = 0.5*log(s2 / p2);					
				else
					lambda = 0.0;

				taux = shdx[q] / shlen[q];
				tauy = shdy[q] / shlen[q];


				skos0x = alpha * tauy + lambda * taux;
				skos0y = -alpha * taux + lambda * tauy;

				if (schemeSwitcher == 2)
				{
				        u1x = 0.5 / shlen[q] * ((px + sx) * taux * taux \
					     + 2.0 * (py + sy) * taux * tauy - (px + sx) * tauy * tauy);

				        u1y = 0.5 / shlen[q] * (-(py + sy) * taux * taux \
					     + 2.0 * (px + sx) * taux * tauy + (py + sy) * tauy * tauy);

					skos1x = alpha * u1y + lambda * u1x - taux;
					skos1y = -alpha * u1x + lambda * u1y - tauy;
				}

				//if ((i == 0) && (j + q < 10))
				//	printf("j+q = %d, gam_c,gamatt_c,qatt_c = (%f, %f, %f), skos = (%f, %f)\n", (int)(j + q), shfreegamma[q], shattgamma[q], shattsource[q], skos0x, skos0y);

				velx += (shfreegamma[q] + shattgamma[q]) * (-skos0y) + shattsource[q] * skos0x;
				vely += (shfreegamma[q] + shattgamma[q]) * ( skos0x) + shattsource[q] * skos0y;

				if (schemeSwitcher == 2)
				{
					//if ((i == 0) && (j + q < 10))
					//	printf("j+q = %d, gam_l,gamatt_l,qatt_l = (%f, %f, %f), skos = (%f, %f)\n", (int)(j + q), shfreegammaLin[q], shattgammaLin[q], shattsourceLin[q], skos1x, skos1y);

					velx += (shfreegammaLin[q] + shattgammaLin[q]) * (-skos1y) + shattsourceLin[q] * skos1x;
					vely += (shfreegammaLin[q] + shattgammaLin[q]) * ( skos1x) + shattsourceLin[q] * skos1y;
				}
			}
		}
		__syncthreads();
	}

	if (i < npt)
	{
		vel[2 * i + 0] = velx * invdpi;
		vel[2 * i + 1] = vely * invdpi;
	}
}


__global__ void CU_calc_I1I2(
	size_t npt, double* pt,
	size_t nvt, double* vt,
	double* i1, double* i2,
	double* rd, double minRd)
{
	__shared__ double shx[CUBLOCK];
	__shared__ double shy[CUBLOCK];
	__shared__ double shg[CUBLOCK];
	
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;

	double ptx = pt[i*sizeVort + posR + 0];
	double pty = pt[i*sizeVort + posR + 1];
	double rdi = myMax(rd[i], minRd);

	double val1 = 0.0;
	double val2x = 0.0;
	double val2y = 0.0;

	double dx, dy, dr;
	double expr, exprdivdr;
	
	double diffRadius = 8.0 * rdi;

	double left = ptx - diffRadius;
	double right = ptx + diffRadius;

	for (size_t j = 0; j < nvt; j += CUBLOCK)
	{
		shx[threadIdx.x] = vt[(j + threadIdx.x)*sizeVort + posR + 0];
		shy[threadIdx.x] = vt[(j + threadIdx.x)*sizeVort + posR + 1];
		shg[threadIdx.x] = vt[(j + threadIdx.x)*sizeVort + posG + 0];
		
		__syncthreads();

		for (size_t q = 0; q < CUBLOCK; ++q)
		{
			if (j + q < nvt)
			{
				if ((shx[q] < right) && (shx[q] > left))
				{
					dx = ptx - shx[q];
					dy = pty - shy[q];

					dr = sqrt(dx*dx + dy * dy);

					if ((dr < diffRadius) && (dr > 1e-10))
					{
						expr = shg[q] * exp(-dr / rdi);
						exprdivdr = expr / dr;
						val1 += expr;
						val2x += exprdivdr * dx;
						val2y += exprdivdr * dy;
					}//if (rij>1e-10)
				}
			}
		}
		__syncthreads();
	}

	//printf("thread = %d, ptx = %f, rd[i] = %f\n", (int)locI, ptx, rd[i]);

	if (i < npt)
	{
		i1[i] = val1;
		i2[2 * i + 0] = val2x;
		i2[2 * i + 1] = val2y;
	}

	//if ((currentStep == 1) && (i == 0))
	//	printf("fromWake:  i = %d, val1 = %f, val2.x = %f, val2.y = %f\n", (int)i, val1, val2x, val2y);
}


__global__ void CU_calc_I1I2mesh(
	size_t npt, double* pt,
	size_t nvt, double* vt,
	double* i1, double* i2,
	double* rd, int* dev_ptr_mesh)
{
	__shared__ double shx[CUBLOCK];
	__shared__ double shy[CUBLOCK];
	__shared__ double shg[CUBLOCK];
	__shared__ int shmshx[CUBLOCK];
	__shared__ int shmshy[CUBLOCK];

	size_t i = blockIdx.x * blockDim.x + threadIdx.x;

	double ptx = pt[i*sizeVort + posR + 0];
	double pty = pt[i*sizeVort + posR + 1];
	double rdi = rd[i];

	double val1 = 0.0;
	double val2x = 0.0;
	double val2y = 0.0;

	double dx, dy, dr;
	double expr, exprdivdr;
	
	double diffRadius = 8.0*rdi;

	int imshx = dev_ptr_mesh[2 * i + 0];
	int imshy = dev_ptr_mesh[2 * i + 1];

	for (size_t j = 0; j < nvt; j += CUBLOCK)
	{
		shx[threadIdx.x] = vt[(j + threadIdx.x)*sizeVort + posR + 0];
		shy[threadIdx.x] = vt[(j + threadIdx.x)*sizeVort + posR + 1];
		shg[threadIdx.x] = vt[(j + threadIdx.x)*sizeVort + posG + 0];

		shmshx[threadIdx.x] = dev_ptr_mesh[(j + threadIdx.x)*2 + 0];
		shmshy[threadIdx.x] = dev_ptr_mesh[(j + threadIdx.x)*2 + 1];

		__syncthreads();

		for (size_t q = 0; q < CUBLOCK; ++q)
		{
			if (j + q < nvt)
			{
				if ((abs(imshx - shmshx[q]) < 25) && (abs(imshy - shmshy[q]) < 25))
				{
					dx = ptx - shx[q];
					dy = pty - shy[q];

					dr = sqrt(dx*dx + dy * dy);

					if ((dr < diffRadius) && (dr > 1e-10))
					{
						expr = shg[q] * exp(-dr / rdi);
						exprdivdr = expr / dr;
						val1 += expr;
						val2x += exprdivdr * dx;
						val2y += exprdivdr * dy;
					}//if (rij>1e-10)
				}
			}
		}
		__syncthreads();
	}

	if (i < npt) 
	{
		i1[i] = val1;
		i2[2 * i + 0] = val2x;
		i2[2 * i + 1] = val2y;
	}
}



__global__ void CU_calc_I1I2FromPanels(
	size_t npt, double* pt,
	size_t npnl, double* r, 
	double* freegamma, double* freegammalin,
	double* i1, double* i2,
	double* rd, double minRd)
{
	__shared__ double shx[CUBLOCK];
	__shared__ double shy[CUBLOCK];
	__shared__ double shxp1[CUBLOCK];
	__shared__ double shyp1[CUBLOCK];

	__shared__ double shtaux[CUBLOCK];
	__shared__ double shtauy[CUBLOCK];

	__shared__ double shlen[CUBLOCK];
	__shared__ double shptG[CUBLOCK];

	size_t i = blockIdx.x * blockDim.x + threadIdx.x;

	double ptx, pty, rdi;

	if (i < npt)
	{
		ptx = pt[i * sizeVort + posR + 0];
		pty = pt[i * sizeVort + posR + 1];
		rdi = myMax(rd[i], minRd);
	}
	double val1 = 0.0;
	double val2x = 0.0;
	double val2y = 0.0;

	double x0, y0, mn;

	double dx, dy, dr;
	double expr, exprdivdr;

	double diffRadius = 8.0*rdi;
	double left = ptx - diffRadius;
	double right = ptx + diffRadius;

	const int nQuadPt = 3; 

	for (size_t j = 0; j < npnl; j += CUBLOCK)
	{
		if (j + threadIdx.x < npnl)
		{
			shx[threadIdx.x] = r[(j + threadIdx.x) * 4 + 0];
			shy[threadIdx.x] = r[(j + threadIdx.x) * 4 + 1];

			shxp1[threadIdx.x] = r[(j + threadIdx.x) * 4 + 2];
			shyp1[threadIdx.x] = r[(j + threadIdx.x) * 4 + 3];

			shtaux[threadIdx.x] = shxp1[threadIdx.x] - shx[threadIdx.x];
			shtauy[threadIdx.x] = shyp1[threadIdx.x] - shy[threadIdx.x];

			shlen[threadIdx.x] = sqrt(shtaux[threadIdx.x] * shtaux[threadIdx.x] + shtauy[threadIdx.x] * shtauy[threadIdx.x]);

			// todo: Учесть freegammalin
			shptG[threadIdx.x] = freegamma[j + threadIdx.x] * shlen[threadIdx.x] / nQuadPt;
		}

		__syncthreads();

		for (size_t q = 0; q < CUBLOCK; ++q)
		{
			if (j + q < npnl)
			{
				for (int s = 0; s < nQuadPt; ++s)
					{
					mn = (s + 0.5) / nQuadPt;
					x0 = shx[q] + shtaux[q] * mn;
					if ((x0 < right) && (x0 > left))
                    {
						y0 = shy[q] + shtauy[q] * mn;


						dx = ptx - x0;
						dy = pty - y0;

						dr = sqrt(dx*dx + dy * dy);

						if ((dr < diffRadius) && (dr > 1e-10))
						{
							expr = shptG[q] * exp(-dr / rdi);
							exprdivdr = expr / dr;
							val1 += expr;
							val2x += exprdivdr * dx;
							val2y += exprdivdr * dy;
						}//if (rij>1e-10)
					}
				}
			}
		}
		__syncthreads();
	}

	if (i < npt)
	{
		i1[i] = val1;
		i2[2 * i + 0] = val2x;
		i2[2 * i + 1] = val2y;
	}

	//if ((currentStep == 1) && (i == 0))
	//	printf("fromSheet: i = %d, val1 = %f, val2.x = %f, val2.y = %f\n", (int)i, val1, val2x, val2y);
}


__global__ void  CU_calc_I0I3(
	size_t npt, double* pt, //вихри
	size_t nvt, double* vt, //панели
	double* i0, double* i3,
	double* rd, double* meanEps,
	double minRd,
	double* visstr)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;	

	double ptx = pt[i*sizeVort + posR + 0];
	double pty = pt[i*sizeVort + posR + 1];
	double ptg = pt[i*sizeVort + posG + 0];
	
	
	double rdi = myMax(rd[i], minRd);   // commented 30-05
	//double rdi = rd[i];

	double val0 = 0.0;
	double val3x = 0.0;
	double val3y = 0.0;

	double iDDomRad = 1.0 / rdi;

	double qx, qy, d/*, q*/;
	double begx, begy, endx, endy;
	double lenj, lenj_m;
	double taux, tauy;
	double s;
	double normx, normy;
	double v0x, v0y;
	double hx, hy;
	double xix, xiy, lxi;
	double expon;
	double mnx, mny;
	int new_n;
	double den;
	double xi_mx, xi_my, lxi_m;
	double mnog1;
	double vs;
	double meanepsj2;

	if (i < npt) //Здесь так делать можно, т.к. shared memory не используется
	{
		for (size_t j = 0; j < nvt; ++j)
		{
			begx = vt[j * 4 + 0];
			begy = vt[j * 4 + 1];
			endx = vt[j * 4 + 2];
			endy = vt[j * 4 + 3];
			
			vs = 0.0;

			qx = ptx - 0.5 * (begx + endx);
			qy = pty - 0.5 * (begy + endy);
			//q = sqrt(qx * qx + qy * qy);

			lenj = sqrt( (endx - begx)*(endx - begx) + (endy - begy)*(endy - begy) );

			taux = (endx - begx) / lenj;
			tauy = (endy - begy) / lenj;

			s = qx * taux + qy * tauy;

			normx = tauy;
			normy = -taux;
					   
			d = fabs(qx*normx + qy * normy);
			
			meanepsj2 = sqr(myMax(meanEps[j], minRd));    // commented 30-05
			//meanepsj2 = sqr(meanEps[j]);

			if ( (d < 50.0 * lenj) && (fabs(s) < 50.0 * lenj) )	//Почему зависит от длины панели???
			{
				v0x = taux * lenj;
				v0y = tauy * lenj;
								
				if ( (d > 5.0 * lenj) || (fabs(s) > 5.0 * lenj) )
				{
					xix = qx * iDDomRad;
					xiy = qy * iDDomRad;
					lxi = sqrt(xix * xix + xiy * xiy);

					expon = exp(-lxi) * lenj;
					mnx = normx * expon;
					mny = normy * expon;

					if (val0 != -pi * rdi)
					{
						val0 += (xix * mnx + xiy * mny) * (lxi + 1.0) / (lxi * lxi);
						val3x += mnx;
						val3y += mny;
					}

					vs = ptg * expon / (pi * meanepsj2);

				}					
				//else if ( (d >= 0 * lenj) || (fabs(s) >= 0 * lenj) )
				else if ( (d >= 0.01 * lenj) || (fabs(s) > 0.45  * lenj) )
				{
					//new_n = 100;
					//new_n = (int)(ceil(5.0 * lenj / q));

					den = (fabs(s) < 0.5 * lenj) ? d : (fabs(s) + d - 0.5 * lenj);
					new_n = (int)myMin((int)myMax(ceil(10.0 * lenj / den), 1.0), 20);

					hx = v0x / new_n;
					hy = v0y / new_n;

					for (int m = 0; m < new_n; ++m)
					{	
						//double midx = ptx - d * normx;
						//double midy = pty - d * normy;
						//double midintx = begx + hx * (m + 0.5);
						//double midinty = begx + hy * (m + 0.5);
						//int dopnew_n = 1;

						//if(((midintx - midx)* (midintx - midx) + (midinty - midy) * (midinty - midy)) < 100*d*d)
						//	dopnew_n = (int)myMin((int)myMax(ceil(1000.0 * lenj / den), 1.0), 2000);
						

						//for (int mm = 0; mm < dopnew_n; ++mm)
						//{
							xi_mx = (ptx - (begx + hx * (m + 0.5))) * iDDomRad;
							xi_my = (pty - (begy + hy * (m + 0.5))) * iDDomRad;

							lxi_m = sqrt(xi_mx * xi_mx + xi_my * xi_my);

							lenj_m = lenj / new_n;
							expon = exp(-lxi_m) * lenj_m;

							mnx = normx * expon;
							mny = normy * expon;

							if (val0 != -pi * rdi)
							{
								val0 += (xi_mx * mnx + xi_my * mny) * (lxi_m + 1.0) / (lxi_m * lxi_m);
								val3x += mnx;
								val3y += mny;
							}

							vs += expon;
						//}//for mm
					}//for m
					vs *= ptg / (pi * meanepsj2);

					/*
					if ((d <= 0.001 * lenj) && (fabs(s) < 0.45 * lenj))
					{
						double Rval0 = -pi * rdi;

						double Rmnog1 = 2.0 * rdi * (1.0 - exp(-lenj * 0.5 * iDDomRad) * cosh(fabs(s) * iDDomRad));
						double Rval3x = Rmnog1 * normx;
						double Rval3y = Rmnog1 * normy;
						double Rvs = Rmnog1 * ptg / (pi * meanepsj2);

						printf("v0: %f <-> %f, v3: (%f, %f) <-> (%f, %f), vs: %f <-> %f\n", val0, Rval0, val3x, val3y, Rval3x, Rval3y, vs, Rvs);
					}
					//*/

				}  				
				else
				{
					val0 = -pi * rdi;
										
					mnog1 = 2.0 * rdi * (1.0 - exp(-lenj * 0.5 * iDDomRad) * cosh(fabs(s) * iDDomRad));
					val3x = mnog1 * normx;
					val3y = mnog1 * normy;
					vs = mnog1 * ptg / (pi * meanepsj2);

				}
				
			}//if d<50 len 
			//if (i == 12)
			//	printf("j = %d, val3: %f, %f\n", (int)j, val3x, val3y);
#ifdef  __CUDA_ARCH__
#if __CUDA_ARCH__ <= 600
			myAtomicAdd(visstr + j, vs);
			//printf("MY: __CUDA_ARCH__ = %d\n", __CUDA_ARCH__);
#else
			atomicAdd(visstr + j, vs);
			//printf("CUDA: __CUDA_ARCH__ = %d\n", __CUDA_ARCH__);
#endif
#endif //  __CUDA_ARCH__			

		}//for j


	}

	if (i < npt)
	{
		i0[i] = val0;
		i3[2 * i + 0] = val3x;
		i3[2 * i + 1] = val3y;
	}
}


__global__ void CU_calc_RHS(	
	size_t npt, double* pt,
	size_t nvt, double* vt,
	size_t nsr, double* sr,
	double eps2,
	double* rhs,
	double* rhsLin)
{
	__shared__ double shx[CUBLOCK];
	__shared__ double shy[CUBLOCK];
	__shared__ double shg[CUBLOCK];

	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	   
	double begx = 0.0, begy = 0.0, endx = 0.0, endy = 0.0, dlen = 0.0, dix = 0.0, diy = 0.0, taux = 0.0, tauy = 0.0, u1x = 0.0, u1y = 0.0;
	if (i < npt)
	{
		begx = pt[i * 4 + 0];
		begy = pt[i * 4 + 1];
		endx = pt[i * 4 + 2];
		endy = pt[i * 4 + 3];

		dix = endx - begx;
		diy = endy - begy;

		dlen = sqrt(dix * dix + diy * diy);

		taux = dix / dlen;
		tauy = diy / dlen;
	}

	double val = 0.0;
	double valLin = 0.0;

	double sx, sy, px, py;
	double alpha, lambda, tempVel, tempVelLin; //из двух к-тов alpha и lambda в принципе можно для экономии пользоваться одной и той же переменной

	//vortices
	for (size_t j = 0; j < nvt; j += CUBLOCK)
	{
		shx[threadIdx.x] = vt[(j + threadIdx.x) * sizeVort + posR + 0];
		shy[threadIdx.x] = vt[(j + threadIdx.x) * sizeVort + posR + 1];
		shg[threadIdx.x] = vt[(j + threadIdx.x) * sizeVort + posG + 0];

		__syncthreads();

		for (size_t q = 0; q < CUBLOCK; ++q)
		{
			if (j + q < nvt)
			{
				if ((schemeSwitcher == 1) || (schemeSwitcher == 2))
				{
					sx = shx[q] - begx;
					sy = shy[q] - begy;

					px = shx[q] - endx;
					py = shy[q] - endy;

					alpha = atan2(px * sy - py * sx, px * sx + py * sy);

					tempVel = shg[q] * alpha;
					val -= tempVel;
					//printf("pnl = %d, val = %f\n", locI, val);
				}

				if (schemeSwitcher == 0)
					val += 0.5*(shg[q] * dlen / myMax(dlen*dlen, eps2)) * (taux * (sx+px) + tauy * (sy+py));					
								
				if (schemeSwitcher == 2)
				{
					u1x = 0.5 / dlen * ((px + sx) * taux * taux \
						+ 2.0 * (py + sy) * taux * tauy - (px + sx) * tauy * tauy);
					u1y = 0.5 / dlen * (-(py + sy) * taux * taux \
						+ 2.0 * (px + sx) * taux * tauy + (py + sy) * tauy * tauy);

					lambda = 0.5 * log((sx * sx + sy * sy) / (px * px + py * py));

					tempVelLin = shg[q] * (alpha * (u1x * taux + u1y * tauy) + lambda * (-u1y * taux + u1x * tauy));

					valLin -= tempVelLin;
				}
			}
		}
		__syncthreads();
	}

	//sources	
	for (size_t j = 0; j < nsr; j += CUBLOCK)
	{
		shx[threadIdx.x] = sr[(j + threadIdx.x) * sizeVort + posR + 0];
		shy[threadIdx.x] = sr[(j + threadIdx.x) * sizeVort + posR + 1];
		shg[threadIdx.x] = sr[(j + threadIdx.x) * sizeVort + posG + 0] * accelCoeff;

		__syncthreads();

		for (size_t q = 0; q < CUBLOCK; ++q)
		{
			if (j + q < nsr)
			{
				if ((schemeSwitcher == 1) || (schemeSwitcher == 2))
				{
					sx = shx[q] - begx;
					sy = shy[q] - begy;

					px = shx[q] - endx;
					py = shy[q] - endy;

					lambda = 0.5 * log((sx * sx + sy * sy) / (px * px + py * py));

					tempVel = shg[q] * lambda;
					val -= tempVel;
				}

				if (schemeSwitcher == 0)
				{
					val += 0.5*(shg[q] * dlen / myMax(dlen*dlen, eps2)) * (-tauy * (sx+px) + taux * (sy+py));
				}

				if (schemeSwitcher == 2)
				{
					u1x = 0.5 / dlen * ((px + sx) * taux * taux \
						+ 2.0 * (py + sy) * taux * tauy - (px + sx) * tauy * tauy);
					u1y = 0.5 / dlen * (-(px + sx) * taux * taux \
						+ 2.0 * (px + sx) * taux * tauy + (py + sy) * tauy * tauy);

					alpha = atan2(px * sy - py * sx, px * sx + py * sy);

					tempVelLin = shg[q] * (alpha * (u1y * taux - u1x * tauy) + lambda * (u1x * taux + u1y * tauy) - 1.0);

					valLin -= tempVelLin;
				}
			}
		}
		__syncthreads();
	}


	if (i < npt)
	{
		val *= invdpi / dlen;
		rhs[i] = val;

		if (schemeSwitcher == 2)
		{
			valLin *= invdpi / dlen;
			rhsLin[i] = valLin;
		}
	}
}


__global__ void CU_calc_mesh(
	size_t npt, double* pt,
	int* dev_ptr_mesh,
	double meshStep)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < npt)
	{
		double ptx = pt[i * sizeVort + posR + 0];
		double pty = pt[i * sizeVort + posR + 1];

		dev_ptr_mesh[2 * i + 0] = floor(ptx / meshStep);
		dev_ptr_mesh[2 * i + 1] = floor(pty / meshStep);
	}
}


__global__ void CU_calc_nei(
	size_t npt, double* pt,
	int* dev_ptr_mesh, int* dev_ptr_nei,
	double epsCol2, int type)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	size_t minI = blockIdx.x * blockDim.x;

	if (i < npt) // можно, т.к. не используем shared memory
	{
		int ix = dev_ptr_mesh[2 * i + 0];
		int iy = dev_ptr_mesh[2 * i + 1];

		int jx, jy;

		double ipx, ipy, jpx, jpy;
		ipx = pt[i*sizeVort + posR + 0];
		ipy = pt[i*sizeVort + posR + 1];

		double dx, dy, r2, r2test;
		dev_ptr_nei[i] = 0;

		double ig = pt[i*sizeVort + posG + 0];
		double jg;

				
		double cftmax = myMax(1.0, /* 2.0 * */ (ipx-collapseRightBorder) / collapseScale);
		//double cftmax = myMax(1.0, (ipx - 0.5) / 0.1);
		
		double cftmax2 = cftmax * cftmax;

		r2test = (type == 1) ? 4.0*epsCol2 * cftmax2 : epsCol2 * cftmax2;
		
		//if (cftmax > 1)
		//	r2test = sqr(0.005*collapseScale);

		int fracMesh = (int)(r2test / epsCol2);
   
		bool cond;

		for (size_t j = minI; j < npt; ++j)
		{
			jx = dev_ptr_mesh[2 * j + 0];
			jy = dev_ptr_mesh[2 * j + 1];

			if ((sqr(abs(ix - jx)) <= fracMesh) && (sqr(abs(iy - jy)) <= fracMesh) && (j > i))
			{
				jpx = pt[j*sizeVort + posR + 0];
				jpy = pt[j*sizeVort + posR + 1];

				dx = ipx - jpx;
				dy = ipy - jpy;

				r2 = dx*dx + dy*dy;
				jg = pt[j*sizeVort + posG + 0];

				cond = (r2 < r2test) && ((type == 1) ? ig*jg < 0 : (   (type == 0) ? ig*jg !=0 : ig*jg > 0  ) && (fabs(ig + jg) < cftmax2 * maxGamma) );
                if (cond)
				{
					dev_ptr_nei[i] = j;
					break;
				}
			}
		}
	}
}

void cuDevice(int n)
{
	hipSetDevice(n);
}


//ниже - обычные (__host__) функции

int cuCalcBlocks(size_t new_n)
{
	size_t nBlocks = new_n / CUBLOCK;
	if (new_n % CUBLOCK)
		nBlocks++;
	return max((int)nBlocks, 1);
}

void cuSetConstants(size_t pos_, size_t posR_, size_t posG_, int code)
{
	hipError_t err1 = hipMemcpyToSymbol(HIP_SYMBOL(sizeVort), &pos_,  sizeof(size_t));
	hipError_t err2 = hipMemcpyToSymbol(HIP_SYMBOL(posR),     &posR_, sizeof(size_t));
	hipError_t err3 = hipMemcpyToSymbol(HIP_SYMBOL(posG),     &posG_, sizeof(size_t));

	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erSetConst01, code = " << code << ")" << std::endl;
	if (err2 != hipSuccess)
		std::cout << hipGetErrorString(err2) << " (erSetConst02, code =" << code << ")" << std::endl;
	if (err3 != hipSuccess)
		std::cout << hipGetErrorString(err3) << " (erSetConst03, code =" << code << ")" << std::endl;
}

void cuSetAccelCoeff(double cft_, int code)
{
	hipError_t err1 = hipMemcpyToSymbol(HIP_SYMBOL(accelCoeff), &cft_, sizeof(double));

	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erSetAccelCoeff01, code =" << code << ")" << std::endl;
}


void cuSetCollapseCoeff(double pos_, double refLength_, int code)
{
	hipError_t err1 = hipMemcpyToSymbol(HIP_SYMBOL(collapseRightBorder), &pos_, sizeof(double));

	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erSetCollapseCoeff01, code =" << code << ")" << std::endl;

	hipError_t err2 = hipMemcpyToSymbol(HIP_SYMBOL(collapseScale), &refLength_, sizeof(double));

	if (err2 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erSetCollapseCoeff02, code =" << code << ")" << std::endl;
}


void cuSetMaxGamma(double gam_, int code)
{
	hipError_t err1 = hipMemcpyToSymbol(HIP_SYMBOL(maxGamma), &gam_, sizeof(double));

	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erSetMaxGamma01, code =" << code << ")" << std::endl;
}

void cuSetSchemeSwitcher(int schemeSwitcher_, int code)
{
	hipError_t err1 = hipMemcpyToSymbol(HIP_SYMBOL(schemeSwitcher), &schemeSwitcher_, sizeof(int));

	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erSchemeSwitcher01, code =" << code << ")" << std::endl;
}

void cuSetCurrentStep(int currentStep_, int code)
{
	hipError_t err1 = hipMemcpyToSymbol(HIP_SYMBOL(currentStep), &currentStep_, sizeof(int));

	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erCurrentStep01, code =" << code << ")" << std::endl;
}



void cuReserveDevMem(void*& ptr, size_t nBytes, int code)
{
	hipError_t err1 = hipMalloc(&ptr, nBytes);
	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erReserveDevMem01, code =" << code << ")" << std::endl;

}


void cuCheckError(int code)
{
	hipDeviceSynchronize();
	hipError_t err1 = hipGetLastError();
	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (cuCheckError) " << code << std::endl;
}


void cuClearWakeMem(size_t new_n, double* dev_ptr, int code)
{
	dim3 blocks(cuCalcBlocks(new_n)), threads(CUBLOCK);

	CU_WakeToZero<<<blocks, threads>>> (new_n, dev_ptr);

	hipError_t err1 = hipGetLastError();
	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erCU_WakeToZero01)" << std::endl;
}

void cuCopyWakeToDev(size_t n, const Vortex2D* host_src, double* dev_ptr, int code)
{
	size_t sizeOfVortex = sizeof(Vortex2D);
	hipError_t err1 = hipMemcpy(dev_ptr, host_src, sizeOfVortex * n, hipMemcpyHostToDevice);
	if (err1 != hipSuccess)
	{
		std::cout << hipGetErrorString(err1) << " (erCopyWakeToDev01, code =" << code << ")" << std::endl;
	}

}

void cuCopyWakeToDevAsync(size_t n, const Vortex2D* host_src, double* dev_ptr, int code)
{
	size_t sizeOfVortex = sizeof(Vortex2D);
	hipError_t err1 = hipMemcpyAsync(dev_ptr, host_src, sizeOfVortex * n, hipMemcpyHostToDevice);
	if (err1 != hipSuccess)
	{
		std::cout << hipGetErrorString(err1) << " (erCopyWakeToDevAsync01, code =" << code << ")" << std::endl;
	}

}


void cuCopyFixedArray(void* dev_ptr, void* host_src, size_t nBytes, int code)
{
	hipError_t err1 = hipMemcpy(dev_ptr, host_src, nBytes, hipMemcpyHostToDevice);
	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erCopyFixedArray01, code =" << code << ")" << std::endl;
}

void cuCopyFixedArrayPoint2D(double* dev_ptr, const Point2D* host_src, size_t npts, int code)
{
	hipError_t err1 = hipMemcpy(dev_ptr, host_src, sizeof(double) * 2 * npts, hipMemcpyHostToDevice);

	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erCopyFixedArrayPoint2D01, code = " << code << ")" << std::endl;
}

void cuCopyFixedArrayPoint4D(double* dev_ptr, const Point2D* host_src, size_t npts, int code)
{
	hipError_t err1 = hipMemcpy(dev_ptr, host_src, sizeof(double) * 4 * npts, hipMemcpyHostToDevice);

	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erCopyFixedArrayPoint4D01, code = " << code << ")" << std::endl;
}

void cuCopyMemFromDev(void* host_ptr, void* dev_ptr, size_t nBytes, int code)
{
	hipError_t err1 = hipMemcpy(host_ptr, dev_ptr, nBytes, hipMemcpyDeviceToHost);
	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erCopyMemFromDev01, code =" << code << ")" << std::endl;
}

void cuDeleteFromDev(void* devPtr, int code)
{
	hipError_t err1 = hipFree(devPtr);

	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erDeleteFromDev01, code =" << code << ")" << std::endl;
}

/////////////////////////////////////////////////////////////
void cuCalculateConvVeloWake(size_t npt, double* pt, size_t nvt, double* vt, size_t nsr, double* sr, size_t nAfls, size_t* nVtxs, double** ptrVtxs, double* vel, double* rd, double eps2, bool calcVelo, bool calcRadius)
{	
	dim3 blocks(cuCalcBlocks(npt)), threads(CUBLOCK);


	/*
	hipEvent_t start, stop;
	float gpu_time = 0.0f;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0); 
	*/

	CU_calc_conv_epsast <<< blocks, threads >>> (npt, pt, nvt, vt, nsr, sr, eps2, vel, rd, nAfls, nVtxs, ptrVtxs, calcVelo, calcRadius);

	/*
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpu_time, start, stop);
	printf("\nTime spent: %.5f\n", gpu_time/1000.0f);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	*/


	hipError_t err1 = hipGetLastError();
	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erCU_calc_conv_epsast01)" << std::endl;

	hipDeviceSynchronize();
}


void cuCalculateConvVeloWakeFromVirtual(size_t npt, double* pt, size_t npnl, double* r, double* freegamma, double* freegammalin, double* attgamma, double* attgammalin, double* attsource, double* attsourcelin, double* vel, double eps2)
{
	dim3 blocks(cuCalcBlocks(npt)), threads(CUBLOCK);
	CU_calc_conv_From_Panels<<<blocks, threads>>>(npt, pt, npnl, r, freegamma, freegammalin, attgamma, attgammalin, attsource, attsourcelin, eps2, vel);

	hipError_t err1 = hipGetLastError();
	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erCU_calc_conv01)" << std::endl;

	hipDeviceSynchronize();
}


void cuCalculateDiffVeloWake(size_t npt, double* pt, size_t nvt, double* vt, double* i1, double* i2, double* rd, double minRad)
{
	dim3 blocks(cuCalcBlocks(npt)), threads(CUBLOCK);
	CU_calc_I1I2 << < blocks, threads >> > (npt, pt, nvt, vt, i1, i2, rd, minRad);

	hipError_t err1 = hipGetLastError();
	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erCU_calc_I1I201)" << std::endl;
}

void cuCalculateDiffVeloWakeMesh(size_t npt, double* pt, size_t nvt, double* vt, int* mesh, double meshStep, double* i1, double* i2, double* rd)
{
	dim3 blocks1(cuCalcBlocks(nvt)), blocks2(cuCalcBlocks(npt)), threads(CUBLOCK);
	CU_calc_mesh << < blocks1, threads >> > (nvt, vt, mesh, meshStep);
	
	hipError_t err1 = hipGetLastError();
	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erCU_calc_mesh01)" << std::endl;

	CU_calc_I1I2mesh << < blocks2, threads >> > (npt, pt, nvt, vt, i1, i2, rd, mesh);

	hipError_t err2 = hipGetLastError();
	if (err2 != hipSuccess)
		std::cout << hipGetErrorString(err2) << " (erCU_calc_I1I2mesh01)" << std::endl;
}

void cuCalculateDiffVeloWakeFromPanels(size_t npt, double* pt, size_t npnl, double* r, double* freegamma, double* freegammalin, double* i1, double* i2, double* rd, double minRad)
{
	dim3 blocks(cuCalcBlocks(npt)), threads(CUBLOCK);
	CU_calc_I1I2FromPanels << < blocks, threads >> > (npt, pt, npnl, r, freegamma, freegammalin, i1, i2, rd, minRad);

	hipError_t err1 = hipGetLastError();
	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erCU_calc_I1I201)" << std::endl;
}


void cuCalculateSurfDiffVeloWake(size_t npt, double* pt, size_t nvt, double* vt, double* i0, double* i3, double* rd, double* meanEps, double minRd, double* visstr)
{
	dim3 blocks(cuCalcBlocks(npt)), threads(CUBLOCK);
	CU_calc_I0I3<<<blocks, threads>>>(npt, pt, nvt, vt, i0, i3, rd, meanEps, minRd, visstr);
	
	hipError_t err1 = hipGetLastError();
	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erCU_calc_I0I31)" << std::endl;
}


void cuCalculateRhs(size_t npt, double* pt, size_t nvt, double* vt, size_t nsr, double* sr, double eps2, double* rhs, double* rhsLin)
{
	dim3 blocks(cuCalcBlocks(npt)), threads(CUBLOCK);
	
	CU_calc_RHS << < blocks, threads >> > (npt, pt, nvt, vt, nsr, sr, eps2, rhs, rhsLin);
	
	hipError_t err1 = hipGetLastError();
	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erCU_calc_RHS01)" << std::endl;
}



void cuCalculatePairs(size_t npt, double* pt, int* mesh, int* nei, double meshStep, double epsCol2, int type)
{
	dim3 blocksMesh(cuCalcBlocks(npt)), threadsMesh(CUBLOCK);
	CU_calc_mesh << < blocksMesh, threadsMesh >> > (npt, pt, mesh, meshStep);

	hipError_t err1 = hipGetLastError();
	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erCU_calc_mesh01)" << std::endl;
	
	dim3 blocksNei(cuCalcBlocks(npt)), threadsNei(CUBLOCK);
	CU_calc_nei << < blocksNei, threadsNei >> > (npt, pt, mesh, nei, epsCol2, type);
	
	hipError_t err2 = hipGetLastError();
	if (err2 != hipSuccess)
		std::cout << hipGetErrorString(err2) << " (erCU_calc_nei01)" << std::endl;
}


/*
void cuTEST(const std::string& str)
{
	double* ppp;
	double mmm = 134;

	hipMalloc(&ppp, 8);

	hipError_t err1 = hipMemcpy(ppp, &mmm, sizeof(double), hipMemcpyHostToDevice);
	if (err1 != hipSuccess)
		std::cout << str << ": " << hipGetErrorString(err1) << " (CUDA_TEST_BREAK)" << std::endl;
	else
		std::cout << str << ": " << hipGetErrorString(err1) << " (CUDA_TEST_PASSED)" << std::endl;
}
*/


void cuInverseMatrix(int n, double* matrPtr, double* invMatrPtr)
{
	hipsolverHandle_t cusolverH = NULL;
	hipStream_t stream = NULL;


	double *devMatr, *devInvMatr; // , * devSol;
	hipMalloc(&devMatr, n * n * sizeof(double));
	hipMalloc(&devInvMatr, n * n * sizeof(double));
	hipError_t cudaStat1 = hipMemcpy(devMatr, matrPtr, sizeof(double) * n * n, hipMemcpyHostToDevice);
	if (cudaStat1 != hipSuccess)
		std::cout << hipGetErrorString(cudaStat1) << " (cuInverseMatrix01)" << std::endl;

	hipError_t cudaStat2 = hipMemcpy(devInvMatr, invMatrPtr, sizeof(double) * n * n, hipMemcpyHostToDevice);
	if (cudaStat2 != hipSuccess)
		std::cout << hipGetErrorString(cudaStat2) << " (cuInverseMatrix02)" << std::endl;

	int* d_Ipiv = NULL; /* pivoting sequence */
	int* d_info = NULL; /* error info */

	int  lwork = 0;     /* size of workspace */
	double* d_work = NULL; /* device workspace for getrf */

	hipError_t cudaStat3 = hipMalloc(&d_Ipiv, sizeof(int) * n);
	if (cudaStat3 != hipSuccess)
		std::cout << hipGetErrorString(cudaStat3) << " (cuInverseMatrix03)" << std::endl;

	hipError_t cudaStat4 = hipMalloc(&d_info, sizeof(int));
	if (cudaStat4 != hipSuccess)
		std::cout << hipGetErrorString(cudaStat4) << " (cuInverseMatrix04)" << std::endl;

	/* step 1: create cusolver handle, bind a stream */
	hipsolverStatus_t status = hipsolverDnCreate(&cusolverH);
	if (HIPSOLVER_STATUS_SUCCESS != status)
		std::cout << " (cuBlas01)" << std::endl;

	hipError_t cudaStat5 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
	if (cudaStat5 != hipSuccess)
		std::cout << hipGetErrorString(cudaStat5) << " (cuInverseMatrix05)" << std::endl;

	status = hipsolverSetStream(cusolverH, stream);
	if (HIPSOLVER_STATUS_SUCCESS != status)
		std::cout << " (cuBlas02)" << std::endl;

	bool pivot_on = true;

	/* step 3: query working space of getrf */
	status = hipsolverDnDgetrf_bufferSize(
		cusolverH,
		n,
		n,
		devMatr,
		n,
		&lwork);
	if (HIPSOLVER_STATUS_SUCCESS != status)
		std::cout << " (cuBlas03)" << std::endl;

	hipError_t cudaStat6 = hipMalloc((void**)&d_work, sizeof(double) * lwork);
	if (cudaStat6 != hipSuccess)
		std::cout << hipGetErrorString(cudaStat6) << " (cuInverseMatrix06)" << std::endl;

	/* step 4: LU factorization */
	if (pivot_on) {
		status = hipsolverDnDgetrf(
			cusolverH,
			n,
			n,
			devMatr,
			n,
			d_work,
			d_Ipiv,
			d_info);
	}
	else {
		status = hipsolverDnDgetrf(
			cusolverH,
			n,
			n,
			devMatr,
			n,
			d_work,
			NULL,
			d_info);
	}
	hipError_t cudaStat7 = hipDeviceSynchronize();

	if (HIPSOLVER_STATUS_SUCCESS != status)
		std::cout << " (cuBlas04)" << std::endl;
	
	if (cudaStat7 != hipSuccess)
		std::cout << hipGetErrorString(cudaStat7) << " (cuInverseMatrix07)" << std::endl;


	/* step 5: solve */
	if (pivot_on) {
		status = hipsolverDnDgetrs(
			cusolverH,
			HIPBLAS_OP_N,
			n,
			n, /* nrhs */
			devMatr,
			n,
			d_Ipiv,
			devInvMatr,
			n,
			d_info);
	}
	else {
		status = hipsolverDnDgetrs(
			cusolverH,
			HIPBLAS_OP_N,
			n,
			1, /* nrhs */
			devMatr,
			n,
			NULL,
			devInvMatr,
			n,
			d_info);
	}

	hipError_t cudaStat8 = hipDeviceSynchronize();

	if (HIPSOLVER_STATUS_SUCCESS != status)
		std::cout << " (cuBlas05)" << std::endl;

	if (cudaStat8 != hipSuccess)
		std::cout << hipGetErrorString(cudaStat8) << " (cuInverseMatrix08)" << std::endl;

	hipError_t cudaStat9 = hipMemcpy(invMatrPtr, devInvMatr, sizeof(double) * n * n, hipMemcpyDeviceToHost);
	if (cudaStat9 != hipSuccess)
		std::cout << hipGetErrorString(cudaStat9) << " (cuInverseMatrix09)" << std::endl;

	hipsolverDnDestroy(cusolverH);
		
	hipError_t cudafree1 = hipFree(devMatr);
	if (cudafree1 != hipSuccess)
		std::cout << hipGetErrorString(cudafree1) << " (cuInverseMatrixFree1)" << std::endl;
	
	hipError_t cudafree2 = hipFree(devInvMatr);
	if (cudafree2 != hipSuccess)
		std::cout << hipGetErrorString(cudafree2) << " (cuInverseMatrixFree2)" << std::endl;
	
	hipError_t cudafree3 = hipFree(d_Ipiv);
	if (cudafree3 != hipSuccess)
		std::cout << hipGetErrorString(cudafree3) << " (cuInverseMatrixFree3)" << std::endl;
	
	hipError_t cudafree4 = hipFree(d_info);
	if (cudafree4 != hipSuccess)
		std::cout << hipGetErrorString(cudafree4) << " (cuInverseMatrixFree4)" << std::endl;
	
	hipError_t cudafree5 = hipFree(d_work);
	if (cudafree5 != hipSuccess)
		std::cout << hipGetErrorString(cudafree5) << " (cuInverseMatrixFree5)" << std::endl;
}